#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// Wprowadzenie funkcji dodawaj�cej dwie zmienne i zapisuj�ce je jako wska�nik
// "__global__" wykonywanie funkcji na ho�cie i przekazywanie do karty graficznej w celu wykonania
__global__ void AddIntsCUDA(int* a, int *b)
{
	a[0] += b[0];
}

int main()
{
	//Podanie zmiennych oraz wska�nik�w kt�re b�d� u�ywane jako zmienne na GPU
	int a = 7, b = 9;
	int *d_a, *d_b;

	// Allokacja pami�ci w GPU
	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	// Kopiowanie z CPU do GPU
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	// Wywo�anie funkcji dodawania ze zmiennymi b�d�cymi w GPU
	// "<<<1, 1>>>" odpowiada za ilo�� w siatce (gridzie) a druga odpowiada za ilo�� w�tk�w w gridzie.
	// "<<<grid, thread>>>" wywo�anie kernela
	AddIntsCUDA<<<1, 1>>>(d_a, d_b);

	// Kopiowaniez GPU do CPU
	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

	// Podanie wyniku
	cout << "Wynik dodawania wynosi " << a << endl;

	// Zwolnienie pami�ci
	hipFree(d_a);
	hipFree(d_b);

	// Zwr�cenie warto�ci
	return 0;
}